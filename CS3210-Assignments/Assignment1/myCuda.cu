#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

#define BLOCK_SIZE 32

typedef struct {
	int width;
	int height;
	float* elements;
	int stride;
}Matrix;


// ============================= Cuda device functions =====================================//


/* allocateMatInDeviceMemory : allocate memory space in device memory with the same size as given matrix */
Matrix allocateMatInDeviceMemory (Matrix A){
	Matrix A_alloc;
	A_alloc.width = A_alloc.stride = A.width;
	A_alloc.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc (&A_alloc.elements, size);

	return A_alloc;
}


/* cpyFromHostToDeviceMemory : copy data from CPU memory to GPU device memory */
void cpyMatFromHostToDevice (Matrix A_host, Matrix A_device){
	size_t size = A_host.width * A_host.height * sizeof(float);
	hipMemcpy (A_device.elements, A_host.elements, size, hipMemcpyHostToDevice);
}

/* cpyMatFromDeviceToHost : copy matrix from device memory to CPU memory */
void cpyMatFromDeviceToHost (Matrix A_device, Matrix A_host){
	size_t size = A_device.width * A_device.height * sizeof(float);
	hipMemcpy (A_host.elements, A_device.elements, size, hipMemcpyDeviceToHost);
}

/* GetElement : get a matrix element */
__device__ float GetElement (const Matrix A, int row, int col){
	return A.elements[row * A.stride + col];
}


/* SetElement : set a matrix element */
__device__ void SetElement (Matrix A, int row, int col, float value){
	A.elements[row * A.stride + col] = value;
}


/* GetSubMatrix : get BLOCK_SIZE by BLOCK_SIZE submatrix of A such that the upper-left corner of A is in row and col of original matrix */
__device__ Matrix GetSubMatrix (Matrix A, int row, int col) {
	
	Matrix A_sub;
	A_sub.width = BLOCK_SIZE;
	A_sub.height = BLOCK_SIZE;
	A_sub.stride = A.stride;
	A_sub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];

	return A_sub;
}


/* mmKernel : multiplication running on GPU core */
__global__ void mmKernel (Matrix A, Matrix B, Matrix C){

	// block row and col
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	// each thread block computes one sub-matrix of C
	Matrix C_sub = GetSubMatrix (C, blockRow, blockCol);

	// accumulating result 
	float Cvalue = 0.0;

	// thread row and col wihtin C_sub
	int row = threadIdx.y;
	int col = threadIdx.x;

	for (int m=0; m<(A.width / BLOCK_SIZE); ++m){
 		// get sub-matrix of A, B from device memory
 		Matrix A_sub = GetSubMatrix (A, blockRow, m);
 		Matrix B_sub = GetSubMatrix (B, m, blockCol);

 		// store sub-matrix of A, B into shared memory
 		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
 		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
 		As[row][col] = GetElement (A_sub, row, col);
 		Bs[row][col] = GetElement (B_sub, row, col);
 		__syncthreads();

 		// compute multiplication
 		for (int e=0; e<BLOCK_SIZE; ++e){
 			Cvalue += As[row][e] * Bs[e][col];
 			__syncthreads();
 		}

 		// write result back to device memory
 		SetElement (C_sub, row, col, Cvalue);
	}
}


/* mm : Matrix Multiplication running on device */
void mmCuda (const Matrix A, const Matrix B, Matrix C){

	// allocate and copy matrix from CPU memory to device memory
	Matrix A_alloc = allocateMatInDeviceMemory (A);
	Matrix B_alloc = allocateMatInDeviceMemory (B);
	Matrix C_alloc = allocateMatInDeviceMemory (C);
	cpyMatFromHostToDevice (A, A_alloc);
	cpyMatFromHostToDevice (B, B_alloc);

	
	dim3 dimBlock (BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid (B.width / dimBlock.x, A.height / dimBlock.y);
	mmKernel<<<dimGrid, dimBlock>>> (A_alloc, B_alloc, C_alloc);
	hipDeviceSynchronize();
	cpyMatFromDeviceToHost(C_alloc, C);

	// free device memory
	hipFree (A_alloc.elements);
	hipFree (B_alloc.elements);
	hipFree (C_alloc.elements);
}

// ============================= CPU functions =====================================//

float HostGetElement (const Matrix A, int row, int col){
	return A.elements[row * A.stride + col];
}

void HostSetElement (Matrix A, int row, int col, float value){
	A.elements[row * A.stride + col] = value;
}

void mm(Matrix A, Matrix B, Matrix C)
{
	int i, j, k;
	int size = A.width;
	float result;

	// Do the multiplication
	for (i = 0; i < size; i++){
		for (j = 0; j < size; j++){
			result = 0.0;
			for(k = 0; k < size; k++){
				result += HostGetElement(A, i, k) * HostGetElement(B, k, j);
			}
			HostSetElement (C, i, j, result);
		}
	}
}

// ============================= Helper Functions =====================================//


/* GetTime : get the clock time */
long long GetTime(){
#ifdef __linux__
	struct timespec tp;
	clock_gettime(CLOCK_REALTIME, &tp);
	return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

/* CheckResult : compare two Matrix */
void CheckResult (Matrix A, Matrix B){
	// Compare the results
	int i, j;
	int correct = 1;
	int size = A.width;

	for (i = 0; correct && i < size; i++)
		for (j = 0; j < size; j++)
			if (HostGetElement(A, i, j) != HostGetElement(B, i, j)) {
				correct = 0;
				break;
			}

	if (correct)
		printf("The result matrices are identical!\n");
	else
		printf("Difference in result matrices at element (%d, %d)!\n", i, j);
}


void work (int size){

	Matrix A, B, C, D;
	long long before, after;

	// Initialize matrix elements
	int i, j;

	A.width = size;
	A.stride = A.height = size;
	A.elements = (float*)malloc(A.width * A.height * sizeof(float));
	B.width = size;
	B.stride = B.height = size;
	B.elements = (float*)malloc(B.width * B.height * sizeof(float));
	C.width = size;
	C.stride = C.height = size;
	C.elements = (float*)malloc(C.width * C.height * sizeof(float));
	D.width = size;
	D.stride = D.height = size;
	D.elements = (float*)malloc(D.width * D.height * sizeof(float));

	for (i = 0; i < size; i++)
		for (j = 0; j < size; j++){
			A.elements[i*A.width + j] = rand() % 10;
			B.elements[i*A.width + j] = rand() % 10;
		}
			

	/*// Perform mm on GPU
	before = GetTime ();
	mm(A, B, D);
	after = GetTime ();
	fprintf(stderr, "Matrix multiplication on CPU took %1.2f seconds\n", ((float)(after - before))/1000000000);
*/
    // Perform mm on GPU
	before = GetTime ();
	mmCuda(A, B, C);
	after = GetTime ();
	//fprintf(stderr, "Matrix multiplication on GPU took %1.2f seconds\n", ((float)(after - before))/1000000000);

	//CheckResult (C, D);
	//print_matrix(C);

}


int main(int argc, char ** argv)
{
	int size;
	srand(0); 
	//printf("Usage: %s <size>\n", argv[0]);
    
	if (argc >= 2)
		size = atoi(argv[1]);
	else
		size = 1024;
		
	//fprintf(stderr,"Sequential matrix multiplication of size %d\n", size);
    
	// Multiply the matrices
	work(size);

	return 0;
}



